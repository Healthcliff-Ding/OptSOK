#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <bits/types/struct_timeval.h>
#include <sys/time.h>
#include "common.h"
#include "common/include/forward_functions.h"
#include "operation/operation_interface.h"

namespace SparseOperationKit {

static void profile_time(size_t id, int step, float cpu_time, float gpu_time) {
  std::cout << "step " << step << ": CPU " 
            << std::fixed << cpu_time / 100 << std::setprecision(9)
            << "ms GPU "  << gpu_time / 100 << "ms" << std::endl; 
}

template <typename EmbeddingType>
__global__ void reorderKernel(const size_t EmbeddingDimension, EmbeddingType const *inputs,
                              uint32_t const *indices, EmbeddingType *outputs, size_t,
                              size_t max_chunk_size, uint32_t const *chunk_sizes) {
  // set indices
  uint32_t gpu_idx = blockIdx.y;
  uint32_t thread_cnt = blockDim.x * blockDim.y;
  uint32_t thread_idx = threadIdx.x + blockDim.x * threadIdx.y;
  uint32_t curr_chunk_size = chunk_sizes[gpu_idx];
  // set shared memory
  extern __shared__ uint32_t idx_smem[];
  // set pointers and offsets
  uint32_t const *curr_input_idx = indices + gpu_idx * max_chunk_size;
  EmbeddingType const *curr_input_emb = inputs + gpu_idx * max_chunk_size * EmbeddingDimension;
  uint32_t size_per_block =
      (curr_chunk_size + gridDim.x * warpSize - 1) / (gridDim.x * warpSize) * warpSize;
  uint32_t lbound = blockIdx.x * size_per_block;
  uint32_t rbound = lbound + size_per_block;
  if (rbound > curr_chunk_size) {
    rbound = curr_chunk_size;
  }
  for (uint32_t offset = lbound; offset < rbound; offset += thread_cnt) {
    uint32_t curr_len = thread_cnt;
    if (offset + curr_len > rbound) {
      curr_len = rbound - offset;
    }
    if (thread_idx < curr_len) {
      idx_smem[thread_idx] = curr_input_idx[offset + thread_idx];
    }
    __syncthreads();
    for (uint32_t warp_idx = threadIdx.y; warp_idx < curr_len; warp_idx += blockDim.y) {
      uint32_t orig_idx = idx_smem[warp_idx];
      uint32_t pos_idx = offset + warp_idx;
      for (uint32_t elem_idx = threadIdx.x; elem_idx < EmbeddingDimension; elem_idx += blockDim.x) {
        outputs[orig_idx * EmbeddingDimension + elem_idx] =
            curr_input_emb[pos_idx * EmbeddingDimension + elem_idx];
      }
    }
    __syncthreads();
  }
}

template <typename EmbeddingType>
__global__ void gatherExKernel(const size_t EmbeddingDimension, EmbeddingType const *inputs,
                               uint32_t const *indices, EmbeddingType *outputs, size_t chunks,
                               size_t max_chunk_size, uint32_t const *chunk_sizes) {
  extern __shared__ uint32_t idx_smem[];
  uint32_t gpu_idx = blockIdx.y;
  uint32_t thread_cnt = blockDim.x * blockDim.y;
  uint32_t thread_idx = threadIdx.x + blockDim.x * threadIdx.y;
  uint32_t curr_chunk_size = chunk_sizes[gpu_idx];
  uint32_t const *curr_input_idx = indices + gpu_idx * max_chunk_size;
  EmbeddingType *curr_output = outputs + gpu_idx * max_chunk_size * EmbeddingDimension;
  uint32_t size_per_block =
      (curr_chunk_size + gridDim.x * warpSize - 1) / (gridDim.x * warpSize) * warpSize;
  uint32_t lbound = blockIdx.x * size_per_block;
  uint32_t rbound = lbound + size_per_block;
  if (rbound > curr_chunk_size) {
    rbound = curr_chunk_size;
  }
  for (uint32_t offset = lbound; offset < rbound; offset += thread_cnt) {
    uint32_t curr_len = thread_cnt;
    if (offset + curr_len > rbound) {
      curr_len = rbound - offset;
    }
    if (thread_idx < curr_len) {
      idx_smem[thread_idx] = curr_input_idx[offset + thread_idx];
    }
    __syncthreads();
    for (uint32_t warp_idx = threadIdx.y; warp_idx < curr_len; warp_idx += blockDim.y) {
      uint32_t pos_idx = offset + warp_idx;
      uint32_t orig_idx = idx_smem[warp_idx];
      for (uint32_t elem_idx = threadIdx.x; elem_idx < EmbeddingDimension; elem_idx += blockDim.x) {
        curr_output[pos_idx * EmbeddingDimension + elem_idx] =
            inputs[orig_idx * EmbeddingDimension + elem_idx];
      }
    }
    __syncthreads();
  }
}

template <typename EmbeddingType>
__global__ static void scatterGradKernel(const size_t EmbeddingDimension, EmbeddingType const *top_grad,
                                  uint32_t const *top_indices, EmbeddingType **replica_grad, 
                                  size_t chunks, size_t max_chunk_size, 
                                  uint32_t const *top_select_size, uint32_t const *replica_recv_offset) {
  uint32_t gpu_idx = blockIdx.y;
  uint32_t curr_chunk_size = top_select_size[gpu_idx];
  uint32_t const *curr_input_idx = top_indices + gpu_idx * max_chunk_size;
  EmbeddingType *curr_output = replica_grad[gpu_idx] + replica_recv_offset[gpu_idx] * EmbeddingDimension;

  for (size_t id = blockIdx.x * blockDim.x + threadIdx.x; id < curr_chunk_size * EmbeddingDimension;
       id += blockDim.x * gridDim.x) {
    size_t item_id = id / EmbeddingDimension;
    size_t embedding_id = id - item_id * EmbeddingDimension;

    size_t index = curr_input_idx[item_id];
    curr_output[id] = top_grad[index * EmbeddingDimension + embedding_id];
  }
}

template <typename ValueType>
class All2AllOutputDispatcher : public Dispatcher {
 public:
  explicit All2AllOutputDispatcher(ConstructionContext_t context)
      : Dispatcher(context),
        resource_mgr_(base_context()->get_resource_mgr()),
        num_keys_per_rank_(base_context()->get_replica_batch_size() *
                           base_context()->get_slot_num() * base_context()->get_nnz_per_slot()) {
    const size_t local_gpu_count = resource_mgr_->get_local_gpu_count();
    h_replica_input_grad_ptr_.reserve(local_gpu_count);
    h_replica_recv_chunk_offset_.reserve(local_gpu_count);

    // Profile Initialize
    for (size_t i = 0; i < 4; ++i) {
      cnt[i] = 0;
      cpu_time_acc[i] = 0.;
      gpu_time_acc[i] = 0.;
    }
  }

  void allocate_forward_spaces() override {}


  void allocate_backward_spaces() override {
    const size_t local_gpu_count = resource_mgr_->get_local_gpu_count();
    const size_t global_gpu_count = resource_mgr_->get_global_gpu_count();
    const size_t embedding_vec_size = base_context()->get_param()->get_embedding_vec_size();
    for (size_t dev_id = 0; dev_id < local_gpu_count; dev_id++) {
      auto &buffer = base_context()->get_buffer(dev_id);
      auto &host_buffer = base_context()->get_host_buffer(dev_id);
      {
        Tensor2<ValueType*> tensor;
        host_buffer->reserve({global_gpu_count}, &tensor);
        h_replica_input_grad_ptr_.push_back(tensor);
      }
      {
        Tensor2<uint32_t> tensor;
        host_buffer->reserve({global_gpu_count}, &tensor);
        h_replica_recv_chunk_offset_.push_back(tensor);
      }
    }  // for dev_id in local_gpu_count
  }

  void forward(const Context_t &replica_context, const bool training) override {}

  void backward(const Context_t &replica_context) override {
    const size_t global_gpu_count = resource_mgr_->get_global_gpu_count();
    const size_t global_replica_id = replica_context->get_global_replica_id();
    const size_t local_replica_id = resource_mgr_->cal_local_id_from_global_id(global_replica_id);
    const auto &local_gpu = resource_mgr_->get_local_gpu(local_replica_id);

    const auto &replica_top_gradients = replica_context->input("replica_top_gradient");
    const auto &replica_selected_indices_buf =
        replica_context->input("replica_selected_indices_buf");
    const auto &replica_num_selected_keys = replica_context->input("replica_num_selected_keys");
    const auto &replica_h_recv_chunk_offsets =
        replica_context->input("replica_h_recv_chunk_offsets");
    const auto &h_num_selected_keys = replica_context->input("replica_h_num_selected_keys");
    const auto &h_num_exchanged_keys = replica_context->input("replica_h_num_exchanged_keys");

    auto &replica_input_grad = replica_context->output("replica_input_grad");

    const size_t embedding_vec_size = base_context()->get_param()->get_embedding_vec_size();

    // Profile Session
    // each thread calls once
    timeval begin, end;
    hipEventCreate(&start[local_replica_id]);
    hipEventCreate(&stop[local_replica_id]);
    float cpu_time, gpu_time;
    cnt[local_replica_id]++;

    //* step 1: issue gradient directly to where it ought to be
    //! assume local_gpu_cnt == global_gpu_cnt 
    //  use CPU thread to gather each peer's ptr
    
    // start profile
    gettimeofday(&begin, 0);

    for (size_t dev_id = 0; dev_id < global_gpu_count; dev_id++) {
      h_replica_input_grad_ptr_[dev_id].get_ptr()[local_replica_id] = 
        replica_input_grad->GetPtrWithType<ValueType>();
      h_replica_recv_chunk_offset_[dev_id].get_ptr()[local_replica_id] =
        replica_h_recv_chunk_offsets->GetPtrWithType<uint32_t>()[dev_id];
    }
    resource_mgr_->sync_cpu_threads();
    //* WRITE style synchronize
    {
      dim3 const grid_dim(2 * local_gpu->get_sm_count() / global_gpu_count, global_gpu_count);
      hipEventRecord(start[local_replica_id], local_gpu->get_stream());
      scatterGradKernel<ValueType><<<grid_dim, 1024ul, 0, local_gpu->get_stream()>>>(
        embedding_vec_size, 
        replica_top_gradients->GetPtrWithType<ValueType>(),
        replica_selected_indices_buf->GetPtrWithType<uint32_t>(),
        h_replica_input_grad_ptr_[local_replica_id].get_ptr(),
        global_gpu_count,
        num_keys_per_rank_,
        replica_num_selected_keys->GetPtrWithType<uint32_t>(),
        h_replica_recv_chunk_offset_[local_replica_id].get_ptr());
      hipEventRecord(stop[local_replica_id], local_gpu->get_stream());
      hipEventSynchronize(stop[local_replica_id]);
      CK_CUDA(hipGetLastError());  
    }
    gettimeofday(&end, 0);
    cpu_time = (1000000.0 * (end.tv_sec - begin.tv_sec) + 
                end.tv_usec - begin.tv_usec) / 1000.0;
    hipEventElapsedTime(&gpu_time, start[local_replica_id], stop[local_replica_id]);
    cpu_time_acc[local_replica_id] += cpu_time;
    gpu_time_acc[local_replica_id] += gpu_time;
    // end profile
    CK_CUDA(hipStreamSynchronize(local_gpu->get_stream()));
    resource_mgr_->sync_cpu_threads();
  
    // Profile Result
    if (cnt[local_replica_id] == 100) {
      auto session_name = []() {
        std::cout << "backward: Output Dispatcher" << std::endl;
      };
      resource_mgr_->blocking_call_once(session_name);
      resource_mgr_->one_at_a_time(profile_time, local_replica_id, 1, 
                                                        cpu_time_acc[local_replica_id], 
                                                        gpu_time_acc[local_replica_id]);
      cpu_time_acc[local_replica_id] = 0.;
      gpu_time_acc[local_replica_id] = 0.;
      cnt[local_replica_id] = 0;
    }
  }

 private:
  std::shared_ptr<ResourcesManager> resource_mgr_;
  const size_t num_keys_per_rank_;

  // forward spaces

  // backward spaces
  Tensors2<ValueType*> h_replica_input_grad_ptr_;
  Tensors2<uint32_t> h_replica_recv_chunk_offset_;

  // profile spaces
  hipEvent_t start[4], stop[4];
  size_t cnt[4];
  float cpu_time_acc[4], gpu_time_acc[4];
};

REGISTER_OUTPUT_DISPATHER_BUILDER("All2AllOutput", DataType::Int64, DataType::Float32,
                                  All2AllOutputDispatcher<float>);
REGISTER_OUTPUT_DISPATHER_BUILDER("All2AllOutput", DataType::Int64, DataType::Float16,
                                  All2AllOutputDispatcher<__half>);
REGISTER_OUTPUT_DISPATHER_BUILDER("All2AllOutput", DataType::Uint32, DataType::Float32,
                                  All2AllOutputDispatcher<float>);
REGISTER_OUTPUT_DISPATHER_BUILDER("All2AllOutput", DataType::Uint32, DataType::Float16,
                                  All2AllOutputDispatcher<__half>);

}  // namespace SparseOperationKit