#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cstddef>
#include <cstdint>
#include <cstring>
#include <string>
#include "common.h"
#include "common/include/forward_functions.h"
#include "operation/operation_interface.h"
#include "tensor_buffer/tensor_interface.h"

namespace SparseOperationKit {

template <typename EmbeddingType>
__global__ void reorderKernel(const size_t EmbeddingDimension, EmbeddingType const *inputs,
                              uint32_t const *indices, EmbeddingType *outputs, size_t,
                              size_t max_chunk_size, uint32_t const *chunk_sizes) {
  // set indices
  uint32_t gpu_idx = blockIdx.y;
  uint32_t thread_cnt = blockDim.x * blockDim.y;
  uint32_t thread_idx = threadIdx.x + blockDim.x * threadIdx.y;
  uint32_t curr_chunk_size = chunk_sizes[gpu_idx];
  // set shared memory
  extern __shared__ uint32_t idx_smem[];
  // set pointers and offsets
  uint32_t const *curr_input_idx = indices + gpu_idx * max_chunk_size;
  EmbeddingType const *curr_input_emb = inputs + gpu_idx * max_chunk_size * EmbeddingDimension;
  uint32_t size_per_block =
      (curr_chunk_size + gridDim.x * warpSize - 1) / (gridDim.x * warpSize) * warpSize;
  uint32_t lbound = blockIdx.x * size_per_block;
  uint32_t rbound = lbound + size_per_block;
  if (rbound > curr_chunk_size) {
    rbound = curr_chunk_size;
  }
  for (uint32_t offset = lbound; offset < rbound; offset += thread_cnt) {
    uint32_t curr_len = thread_cnt;
    if (offset + curr_len > rbound) {
      curr_len = rbound - offset;
    }
    if (thread_idx < curr_len) {
      idx_smem[thread_idx] = curr_input_idx[offset + thread_idx];
    }
    __syncthreads();
    for (uint32_t warp_idx = threadIdx.y; warp_idx < curr_len; warp_idx += blockDim.y) {
      uint32_t orig_idx = idx_smem[warp_idx];
      uint32_t pos_idx = offset + warp_idx;
      for (uint32_t elem_idx = threadIdx.x; elem_idx < EmbeddingDimension; elem_idx += blockDim.x) {
        outputs[orig_idx * EmbeddingDimension + elem_idx] =
            curr_input_emb[pos_idx * EmbeddingDimension + elem_idx];
      }
    }
    __syncthreads();
  }
}

template <typename EmbeddingType>
__global__ void gatherExKernel(const size_t EmbeddingDimension, EmbeddingType const *inputs,
                               uint32_t const *indices, EmbeddingType *outputs, size_t chunks,
                               size_t max_chunk_size, uint32_t const *chunk_sizes) {
  extern __shared__ uint32_t idx_smem[];
  uint32_t gpu_idx = blockIdx.y;
  uint32_t thread_cnt = blockDim.x * blockDim.y;
  uint32_t thread_idx = threadIdx.x + blockDim.x * threadIdx.y;
  uint32_t curr_chunk_size = chunk_sizes[gpu_idx];
  uint32_t const *curr_input_idx = indices + gpu_idx * max_chunk_size;
  EmbeddingType *curr_output = outputs + gpu_idx * max_chunk_size * EmbeddingDimension;
  uint32_t size_per_block =
      (curr_chunk_size + gridDim.x * warpSize - 1) / (gridDim.x * warpSize) * warpSize;
  uint32_t lbound = blockIdx.x * size_per_block;
  uint32_t rbound = lbound + size_per_block;
  if (rbound > curr_chunk_size) {
    rbound = curr_chunk_size;
  }
  for (uint32_t offset = lbound; offset < rbound; offset += thread_cnt) {
    uint32_t curr_len = thread_cnt;
    if (offset + curr_len > rbound) {
      curr_len = rbound - offset;
    }
    if (thread_idx < curr_len) {
      idx_smem[thread_idx] = curr_input_idx[offset + thread_idx];
    }
    __syncthreads();
    for (uint32_t warp_idx = threadIdx.y; warp_idx < curr_len; warp_idx += blockDim.y) {
      uint32_t pos_idx = offset + warp_idx;
      uint32_t orig_idx = idx_smem[warp_idx];
      for (uint32_t elem_idx = threadIdx.x; elem_idx < EmbeddingDimension; elem_idx += blockDim.x) {
        curr_output[pos_idx * EmbeddingDimension + elem_idx] =
            inputs[orig_idx * EmbeddingDimension + elem_idx];
      }
    }
    __syncthreads();
  }
}

template <typename EmbeddingType>
__global__ void scatterGradKernel(const size_t EmbeddingDimension, EmbeddingType const *top_grad,
                                  uint32_t const *top_indices, EmbeddingType **replica_grad, 
                                  size_t chunks, size_t max_chunk_size, 
                                  uint32_t const *top_select_size, uint32_t const *replica_recv_offset) {
  uint32_t gpu_idx = blockIdx.y;
  uint32_t curr_chunk_size = top_select_size[gpu_idx];
  uint32_t const *curr_input_idx = top_indices + gpu_idx * max_chunk_size;
  EmbeddingType *curr_output = replica_grad[gpu_idx] + replica_recv_offset[gpu_idx] * EmbeddingDimension;

  for (size_t id = blockIdx.x * blockDim.x + threadIdx.x; id < curr_chunk_size * EmbeddingDimension;
       id += blockDim.x * gridDim.x) {
    size_t item_id = id / EmbeddingDimension;
    size_t embedding_id = id - item_id * EmbeddingDimension;

    size_t index = curr_input_idx[item_id];
    curr_output[id] = top_grad[index * EmbeddingDimension + embedding_id];
  }
}

template <typename EmbeddingType>
__host__ int embGatherCmp(const EmbeddingType *p2p, const EmbeddingType *nccl, size_t nElem) {
  int res = 0;
  for (size_t i = 0; i < nElem; ++i) {
    if (p2p[i] != nccl[i]) res++;
  }
  return res;
}

template <typename ValueType>
class All2AllOutputDispatcher : public Dispatcher {
 public:
  explicit All2AllOutputDispatcher(ConstructionContext_t context)
      : Dispatcher(context),
        resource_mgr_(base_context()->get_resource_mgr()),
        num_keys_per_rank_(base_context()->get_replica_batch_size() *
                           base_context()->get_slot_num() * base_context()->get_nnz_per_slot()) {
    const size_t local_gpu_count = resource_mgr_->get_local_gpu_count();
    exchanged_embeddings_buf_.reserve(local_gpu_count);
    gathered_gradients_buf_.reserve(local_gpu_count);
    h_output_nccl_.reserve(local_gpu_count);
    h_output_p2p_.reserve(local_gpu_count);
    h_replica_input_grad_ptr_.reserve(local_gpu_count);
    h_replica_recv_chunk_offset_.reserve(local_gpu_count);
    h_grad_nccl_.reserve(local_gpu_count);
    h_grad_p2p_.reserve(local_gpu_count);
  }

  void allocate_forward_spaces() override {
    const size_t local_gpu_count = resource_mgr_->get_local_gpu_count();
    const size_t global_gpu_count = resource_mgr_->get_global_gpu_count();
    const size_t embedding_vec_size = base_context()->get_param()->get_embedding_vec_size();
    for (size_t dev_id = 0; dev_id < local_gpu_count; dev_id++) {
      auto &buffer = base_context()->get_buffer(dev_id);
      auto &host_buffer = base_context()->get_host_buffer(dev_id);
      {
        Tensor2<ValueType> tensor;
        buffer->reserve({global_gpu_count, embedding_vec_size * num_keys_per_rank_}, &tensor);
        exchanged_embeddings_buf_.push_back(tensor);
      }
      {
        Tensor2<ValueType> tensor;
        host_buffer->reserve({global_gpu_count, embedding_vec_size * num_keys_per_rank_}, &tensor);
        h_output_nccl_.push_back(tensor);
      }
      {
        Tensor2<ValueType> tensor;
        host_buffer->reserve({global_gpu_count, embedding_vec_size * num_keys_per_rank_}, &tensor);
        h_output_p2p_.push_back(tensor);
      }
    }  // for dev_id in local_gpu_count
  }

  void allocate_backward_spaces() override {
    const size_t local_gpu_count = resource_mgr_->get_local_gpu_count();
    const size_t global_gpu_count = resource_mgr_->get_global_gpu_count();
    const size_t embedding_vec_size = base_context()->get_param()->get_embedding_vec_size();
    for (size_t dev_id = 0; dev_id < local_gpu_count; dev_id++) {
      auto &buffer = base_context()->get_buffer(dev_id);
      auto &host_buffer = base_context()->get_host_buffer(dev_id);
      {
        Tensor2<ValueType> tensor;  // FIXME: check whether top-grad is fp32 or fp16
        buffer->reserve({global_gpu_count, embedding_vec_size * num_keys_per_rank_}, &tensor);
        gathered_gradients_buf_.push_back(tensor);
      }
      {
        Tensor2<ValueType> tensor;
        host_buffer->reserve({global_gpu_count * embedding_vec_size * num_keys_per_rank_}, &tensor);
        h_grad_nccl_.push_back(tensor);
      }
      {
        Tensor2<ValueType> tensor;
        host_buffer->reserve({global_gpu_count * embedding_vec_size * num_keys_per_rank_}, &tensor);
        h_grad_p2p_.push_back(tensor);
      }
      {
        Tensor2<ValueType*> tensor;
        host_buffer->reserve({global_gpu_count}, &tensor);
        h_replica_input_grad_ptr_.push_back(tensor);
      }
      {
        Tensor2<uint32_t> tensor;
        host_buffer->reserve({global_gpu_count}, &tensor);
        h_replica_recv_chunk_offset_.push_back(tensor);
      }
    }  // for dev_id in local_gpu_count
  }

  void forward(const Context_t &replica_context, const bool training) override {
    const size_t global_gpu_count = resource_mgr_->get_global_gpu_count();
    const size_t global_replica_id = replica_context->get_global_replica_id();
    const size_t local_replica_id = resource_mgr_->cal_local_id_from_global_id(global_replica_id);
    const auto &local_gpu = resource_mgr_->get_local_gpu(local_replica_id);

    const auto &replica_gathered_embeddings = replica_context->input("replica_gathered_embeddings");
    const auto &h_recv_chunk_offsets = replica_context->input("replica_h_recv_chunk_offsets");
    const auto &h_num_exchanged_keys = replica_context->input("replica_h_num_exchanged_keys");
    const auto &h_num_selected_keys = replica_context->input("replica_h_num_selected_keys");
    const auto &replica_num_selected_keys = replica_context->input("replica_num_selected_keys");
    const auto &replica_selected_indices_buf =
        replica_context->input("replica_selected_indices_buf");

    const auto &input_keys = replica_context->input("replica_values");
    auto &replica_output = replica_context->output("replica_output");
    const size_t embedding_vec_size = base_context()->get_param()->get_embedding_vec_size();
    //* P2P Impl
    {
      // P2P has already GATHERed embedding vector
      hipMemcpyAsync(h_output_p2p_[local_replica_id].get_ptr(), 
                      replica_output->GetPtrWithType<ValueType>(),
                      input_keys->get_num_elements() * embedding_vec_size * sizeof(ValueType), 
                      hipMemcpyDeviceToHost,
                      local_gpu->get_stream());
      hipStreamSynchronize(local_gpu->get_stream());
      // MESSAGE("[INFO]: gpu" + std::to_string(local_replica_id) + " finishes P2P Impl.");
      CK_CUDA(hipGetLastError());
    }

    //* NCCL Impl
    // step 1: exchange embedding values among all GPUs.
    CK_NCCL(ncclGroupStart());
    for (size_t dev_id = 0; dev_id < global_gpu_count; dev_id++) {
      CK_NCCL(ncclSend(
          replica_gathered_embeddings->GetPtrWithType<ValueType>() +
              h_recv_chunk_offsets->GetPtrWithType<uint32_t>()[dev_id] * embedding_vec_size,
          h_num_exchanged_keys->GetPtrWithType<uint32_t>()[dev_id] * embedding_vec_size,
          GetNCCLType<ValueType>(), /*peer=*/dev_id, local_gpu->get_nccl(),
          local_gpu->get_stream()));
      CK_NCCL(ncclRecv(exchanged_embeddings_buf_[local_replica_id].get_ptr() +
                           dev_id * num_keys_per_rank_ * embedding_vec_size,
                       h_num_selected_keys->GetPtrWithType<uint32_t>()[dev_id] * embedding_vec_size,
                       GetNCCLType<ValueType>(), /*peer=*/dev_id, local_gpu->get_nccl(),
                       local_gpu->get_stream()));
    }  // for dev_id in global_gpu_count
    CK_NCCL(ncclGroupEnd());

    // step 2: reorder embedding values
    {
      // CK_CUDA(hipMemsetAsync(replica_output->GetPtrWithType<float>(), 0,
      //                         replica_output->get_size_in_bytes(),
      //                         local_gpu->get_stream()));  // TODO: merge it to reorderKernel
      const size_t smem_size = local_gpu->get_max_smem_size_per_sm() / 2;
      CK_CUDA(hipFuncSetAttribute(reinterpret_cast<const void*>(reorderKernel<ValueType>),
                                   hipFuncAttributeMaxDynamicSharedMemorySize, smem_size));
      dim3 const grid_dim(2 * local_gpu->get_sm_count() / global_gpu_count, global_gpu_count);
      dim3 const block_dim(local_gpu->get_warp_size(), EMB_WARPS_PER_BLOCK);
      reorderKernel<ValueType><<<grid_dim, block_dim, smem_size, local_gpu->get_stream()>>>(
          /*EmbeddingDimension=*/embedding_vec_size,
          /*inputs=*/exchanged_embeddings_buf_[local_replica_id].get_ptr(),
          /*indices=*/replica_selected_indices_buf->GetPtrWithType<uint32_t>(),
          /*outputs=*/replica_output->GetPtrWithType<ValueType>(),
          /*chunks=*/global_gpu_count,
          /*max_chunk_size=*/num_keys_per_rank_,
          /*chunk_sizes=*/replica_num_selected_keys->GetPtrWithType<uint32_t>());
      CK_CUDA(hipGetLastError());
    }

    //* TEST
    {
      hipMemcpyAsync(h_output_nccl_[local_replica_id].get_ptr(), 
                      replica_output->GetPtrWithType<ValueType>(),
                      input_keys->get_num_elements() * embedding_vec_size * sizeof(ValueType), 
                      hipMemcpyDeviceToHost,
                      local_gpu->get_stream());
      hipStreamSynchronize(local_gpu->get_stream());
      CK_CUDA(hipGetLastError());
      // MESSAGE("[INFO]: gpu" + std::to_string(local_replica_id) + " ready.");
      // std::cout << (float)h_output_nccl_[local_replica_id].get_ptr()[0] << (float)h_output_p2p_[local_replica_id].get_ptr()[0] << std::endl;
      resource_mgr_->sync_cpu_threads();
      int res = embGatherCmp(h_output_p2p_[local_replica_id].get_ptr(), 
                            h_output_nccl_[local_replica_id].get_ptr(),
                           input_keys->get_num_elements() * embedding_vec_size);
      resource_mgr_->sync_cpu_threads();
      if (res != 0) {
        MESSAGE("[ERROR]: embedding vector in gpu" + std::to_string(local_replica_id) + " is inconsistency");
      }
    }
  }

  void backward(const Context_t &replica_context) override {
    const size_t global_gpu_count = resource_mgr_->get_global_gpu_count();
    const size_t global_replica_id = replica_context->get_global_replica_id();
    const size_t local_replica_id = resource_mgr_->cal_local_id_from_global_id(global_replica_id);
    const auto &local_gpu = resource_mgr_->get_local_gpu(local_replica_id);

    const auto &replica_top_gradients = replica_context->input("replica_top_gradient");
    const auto &replica_selected_indices_buf =
        replica_context->input("replica_selected_indices_buf");
    const auto &replica_num_selected_keys = replica_context->input("replica_num_selected_keys");
    const auto &replica_h_recv_chunk_offsets =
        replica_context->input("replica_h_recv_chunk_offsets");
    const uint32_t h_local_nnz = 
        replica_h_recv_chunk_offsets->GetPtrWithType<uint32_t>()[global_gpu_count];
    const auto &h_num_selected_keys = replica_context->input("replica_h_num_selected_keys");
    const auto &h_num_exchanged_keys = replica_context->input("replica_h_num_exchanged_keys");

    auto &replica_input_grad = replica_context->output("replica_input_grad");

    //* NCCL Impl
    // step 1: gather top gradients for local GPU.
    const size_t embedding_vec_size = base_context()->get_param()->get_embedding_vec_size();
    {
      const size_t smem_size = local_gpu->get_max_smem_size_per_sm() / 2;
      CK_CUDA(hipFuncSetAttribute(reinterpret_cast<const void*>(gatherExKernel<ValueType>),
                                   hipFuncAttributeMaxDynamicSharedMemorySize, smem_size));
      dim3 const grid_dim(2 * local_gpu->get_sm_count() / global_gpu_count, global_gpu_count);
      dim3 const block_dim(local_gpu->get_warp_size(), EMB_WARPS_PER_BLOCK);
      gatherExKernel<ValueType><<<grid_dim, block_dim, smem_size, local_gpu->get_stream()>>>(
          /*EmbeddingDimension=*/embedding_vec_size,
          /*inputs=*/replica_top_gradients->GetPtrWithType<ValueType>(),
          /*indices=*/replica_selected_indices_buf->GetPtrWithType<uint32_t>(),
          /*outputs=*/gathered_gradients_buf_[local_replica_id].get_ptr(),
          /*chunks=*/global_gpu_count,
          /*max_chunk_size=*/num_keys_per_rank_,
          /*chunk_sizes=*/replica_num_selected_keys->GetPtrWithType<uint32_t>());
      CK_CUDA(hipGetLastError());
    }

    // step 2: exchange gradients among all GPUs.
    CK_NCCL(ncclGroupStart());
    for (size_t dev_id = 0; dev_id < global_gpu_count; dev_id++) {
      CK_NCCL(ncclSend(gathered_gradients_buf_[local_replica_id].get_ptr() +
                           dev_id * num_keys_per_rank_ * embedding_vec_size,
                       h_num_selected_keys->GetPtrWithType<uint32_t>()[dev_id] * embedding_vec_size,
                       GetNCCLType<ValueType>(), /*peer=*/dev_id, local_gpu->get_nccl(),
                       local_gpu->get_stream()));
      CK_NCCL(ncclRecv(
          replica_input_grad->GetPtrWithType<ValueType>() +
              replica_h_recv_chunk_offsets->GetPtrWithType<uint32_t>()[dev_id] * embedding_vec_size,
          h_num_exchanged_keys->GetPtrWithType<uint32_t>()[dev_id] * embedding_vec_size,
          GetNCCLType<ValueType>(), /*peer=*/dev_id, local_gpu->get_nccl(),
          local_gpu->get_stream()));
    }  // for dev_id in global_gpu_count
    CK_NCCL(ncclGroupEnd());
    CK_CUDA(hipMemcpyAsync(h_grad_nccl_[local_replica_id].get_ptr(),
                            replica_input_grad->GetPtrWithType<ValueType>(),
                            h_local_nnz * embedding_vec_size * sizeof(ValueType),
                            hipMemcpyDeviceToHost,
                            local_gpu->get_stream()));
    CK_CUDA(hipStreamSynchronize(local_gpu->get_stream()));

    //* P2P Impl
    {
      for (size_t dev_id = 0; dev_id < global_gpu_count; dev_id++) {
        h_replica_input_grad_ptr_[dev_id].get_ptr()[local_replica_id] = 
          replica_input_grad->GetPtrWithType<ValueType>();
        h_replica_recv_chunk_offset_[dev_id].get_ptr()[local_replica_id] =
          replica_h_recv_chunk_offsets->GetPtrWithType<uint32_t>()[dev_id];
      }
      resource_mgr_->sync_cpu_threads();
      //* WRITE style synchronize
      {
        dim3 const grid_dim(2 * local_gpu->get_sm_count() / global_gpu_count, global_gpu_count);
        scatterGradKernel<ValueType><<<grid_dim, 1024ul, 0, local_gpu->get_stream()>>>(
          embedding_vec_size, 
          replica_top_gradients->GetPtrWithType<ValueType>(),
          replica_selected_indices_buf->GetPtrWithType<uint32_t>(),
          h_replica_input_grad_ptr_[local_replica_id].get_ptr(),
          global_gpu_count,
          num_keys_per_rank_,
          replica_num_selected_keys->GetPtrWithType<uint32_t>(),
          h_replica_recv_chunk_offset_[local_replica_id].get_ptr());
        CK_CUDA(hipGetLastError());
      }
      CK_CUDA(hipStreamSynchronize(local_gpu->get_stream()));
      resource_mgr_->sync_cpu_threads();
    }

    //* TEST
    {
      CK_CUDA(hipMemcpyAsync(h_grad_p2p_[local_replica_id].get_ptr(),
                              replica_input_grad->GetPtrWithType<ValueType>(),
                              h_local_nnz * embedding_vec_size * sizeof(ValueType),
                              hipMemcpyDeviceToHost,
                              local_gpu->get_stream()));
      CK_CUDA(hipStreamSynchronize(local_gpu->get_stream()));
      int res = embGatherCmp(h_grad_p2p_[local_replica_id].get_ptr(), 
                             h_grad_nccl_[local_replica_id].get_ptr(),
                             h_local_nnz * embedding_vec_size);
      if (res != 0) {
        MESSAGE("[ERROR]: embedding gradient in gpu" + std::to_string(local_replica_id) + " is inconsistency");
      }
    }
  }

 private:
  std::shared_ptr<ResourcesManager> resource_mgr_;
  const size_t num_keys_per_rank_;

  // forward spaces
  Tensors2<ValueType> exchanged_embeddings_buf_;

  // backward spaces
  Tensors2<ValueType> gathered_gradients_buf_;
  // p2p backward spaces
  Tensors2<ValueType*> h_replica_input_grad_ptr_;
  Tensors2<uint32_t> h_replica_recv_chunk_offset_;

  // test spaces
  Tensors2<ValueType> h_output_nccl_;
  Tensors2<ValueType> h_output_p2p_;
  Tensors2<ValueType> h_grad_nccl_;
  Tensors2<ValueType> h_grad_p2p_;
};

REGISTER_OUTPUT_DISPATHER_BUILDER("All2AllOutput", DataType::Int64, DataType::Float32,
                                  All2AllOutputDispatcher<float>);
REGISTER_OUTPUT_DISPATHER_BUILDER("All2AllOutput", DataType::Int64, DataType::Float16,
                                  All2AllOutputDispatcher<__half>);
REGISTER_OUTPUT_DISPATHER_BUILDER("All2AllOutput", DataType::Uint32, DataType::Float32,
                                  All2AllOutputDispatcher<float>);
REGISTER_OUTPUT_DISPATHER_BUILDER("All2AllOutput", DataType::Uint32, DataType::Float16,
                                  All2AllOutputDispatcher<__half>);

}  // namespace SparseOperationKit